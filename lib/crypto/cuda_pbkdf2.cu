#include "hip/hip_runtime.h"
extern "C"{
#include "cuda_pbkdf2.h"
#include "../utils.h"
}
#include <openssl/hmac.h>
#include <openssl/sha.h>
#include <string.h>
#include <hip/hip_runtime.h>

__device__
void sha1_process( const SHA_DEV_CTX *ctx, SHA_DEV_CTX *data) {

  uint32_t temp, W[80], A, B, C, D, E, i;

  W[ 0] = data->h0;
  W[ 1] = data->h1;
  W[ 2] = data->h2;
  W[ 3] = data->h3;
  W[ 4] = data->h4;
  W[ 5] = 0x80000000;
  W[ 6] = 0;
  W[ 7] = 0;
  W[ 8] = 0;
  W[ 9] = 0;
  W[10] = 0;
  W[11] = 0;
  W[12] = 0;
  W[13] = 0;
  W[14] = 0;
  W[15] = (64+20)*8;

  A = ctx->h0;
  B = ctx->h1;
  C = ctx->h2;
  D = ctx->h3;
  E = ctx->h4;

#undef S
#define S(x,n) ((x << n) | (x >> (32 - n)))

#undef R
#define R(t)                           \
{                                      \
    temp = W[t -  3] ^ W[t -  8] ^     \
           W[t - 14] ^ W[t - 16];      \
           W[t] = S(temp,1);           \
}

#undef P
#define P(a,b,c,d,e,x)                                  \
{                                                       \
    e += S(a,5) + F(b,c,d) + K + x; b = S(b,30);        \
}

#define F(x,y,z) (z ^ (x & (y ^ z)))
#define K 0x5A827999

  for(i=16;i<80;i++)
	  R(i);
  
  
  P( A, B, C, D, E, W[0]  );
  P( E, A, B, C, D, W[1]  );
  P( D, E, A, B, C, W[2]  );
  P( C, D, E, A, B, W[3]  );
  P( B, C, D, E, A, W[4]  );
  P( A, B, C, D, E, W[5]  );
  P( E, A, B, C, D, W[6]  );
  P( D, E, A, B, C, W[7]  );
  P( C, D, E, A, B, W[8]  );
  P( B, C, D, E, A, W[9]  );
  P( A, B, C, D, E, W[10] );
  P( E, A, B, C, D, W[11] );
  P( D, E, A, B, C, W[12] );
  P( C, D, E, A, B, W[13] );
  P( B, C, D, E, A, W[14] );
  P( A, B, C, D, E, W[15] );
  P( E, A, B, C, D, W[16] );
  P( D, E, A, B, C, W[17] );
  P( C, D, E, A, B, W[18] );
  P( B, C, D, E, A, W[19] );
  
#undef K
#undef F

#define F(x,y,z) (x ^ y ^ z)
#define K 0x6ED9EBA1
  
  P( A, B, C, D, E, W[20] );
  P( E, A, B, C, D, W[21] );
  P( D, E, A, B, C, W[22] );
  P( C, D, E, A, B, W[23] );
  P( B, C, D, E, A, W[24] );
  P( A, B, C, D, E, W[25] );
  P( E, A, B, C, D, W[26] );
  P( D, E, A, B, C, W[27] );
  P( C, D, E, A, B, W[28] );
  P( B, C, D, E, A, W[29] );
  P( A, B, C, D, E, W[30] );
  P( E, A, B, C, D, W[31] );
  P( D, E, A, B, C, W[32] );
  P( C, D, E, A, B, W[33] );
  P( B, C, D, E, A, W[34] );
  P( A, B, C, D, E, W[35] );
  P( E, A, B, C, D, W[36] );
  P( D, E, A, B, C, W[37] );
  P( C, D, E, A, B, W[38] );
  P( B, C, D, E, A, W[39] );
  
#undef K
#undef F
  
#define F(x,y,z) ((x & y) | (z & (x | y)))
#define K 0x8F1BBCDC
  
  P( A, B, C, D, E, W[40] );
  P( E, A, B, C, D, W[41] );
  P( D, E, A, B, C, W[42] );
  P( C, D, E, A, B, W[43] );
  P( B, C, D, E, A, W[44] );
  P( A, B, C, D, E, W[45] );
  P( E, A, B, C, D, W[46] );
  P( D, E, A, B, C, W[47] );
  P( C, D, E, A, B, W[48] );
  P( B, C, D, E, A, W[49] );
  P( A, B, C, D, E, W[50] );
  P( E, A, B, C, D, W[51] );
  P( D, E, A, B, C, W[52] );
  P( C, D, E, A, B, W[53] );
  P( B, C, D, E, A, W[54] );
  P( A, B, C, D, E, W[55] );
  P( E, A, B, C, D, W[56] );
  P( D, E, A, B, C, W[57] );
  P( C, D, E, A, B, W[58] );
  P( B, C, D, E, A, W[59] );
  
#undef K
#undef F

#define F(x,y,z) (x ^ y ^ z)
#define K 0xCA62C1D6
  
  P( A, B, C, D, E, W[60] );
  P( E, A, B, C, D, W[61] );
  P( D, E, A, B, C, W[62] );
  P( C, D, E, A, B, W[63] );
  P( B, C, D, E, A, W[64] );
  P( A, B, C, D, E, W[65] );
  P( E, A, B, C, D, W[66] );
  P( D, E, A, B, C, W[67] );
  P( C, D, E, A, B, W[68] );
  P( B, C, D, E, A, W[69] );
  P( A, B, C, D, E, W[70] );
  P( E, A, B, C, D, W[71] );
  P( D, E, A, B, C, W[72] );
  P( C, D, E, A, B, W[73] );
  P( B, C, D, E, A, W[74] );
  P( A, B, C, D, E, W[75] );
  P( E, A, B, C, D, W[76] );
  P( D, E, A, B, C, W[77] );
  P( C, D, E, A, B, W[78] );
  P( B, C, D, E, A, W[79] );
  
#undef K
#undef F

  data->h0 = ctx->h0 + A;
  data->h1 = ctx->h1 + B;
  data->h2 = ctx->h2 + C;
  data->h3 = ctx->h3 + D;
  data->h4 = ctx->h4 + E;

}

__global__ void kernel_pbkdf2_sha1_32( gpu_inbuffer *inbuffer, 
									gpu_outbuffer *outbuffer, int iterations) {
    int i;
	SHA_DEV_CTX temp_ctx, pmk_ctx;
    
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;  
    
    CPY_DEVCTX(inbuffer[idx].e1, temp_ctx);
    CPY_DEVCTX(temp_ctx, pmk_ctx);

    for( i = 0; i < iterations-1; i++ ){
        sha1_process( &inbuffer[idx].ctx_ipad, &temp_ctx);
        sha1_process( &inbuffer[idx].ctx_opad, &temp_ctx);
        pmk_ctx.h0 ^= temp_ctx.h0; pmk_ctx.h1 ^= temp_ctx.h1;
        pmk_ctx.h2 ^= temp_ctx.h2; pmk_ctx.h3 ^= temp_ctx.h3;
        pmk_ctx.h4 ^= temp_ctx.h4;
    }

    CPY_DEVCTX(pmk_ctx, outbuffer[idx].pmk1);
    CPY_DEVCTX(inbuffer[idx].e2, temp_ctx);
    CPY_DEVCTX(temp_ctx, pmk_ctx);

    for( i = 0; i < iterations-1; i++ ){
        sha1_process( &inbuffer[idx].ctx_ipad, &temp_ctx);
        sha1_process( &inbuffer[idx].ctx_opad, &temp_ctx);
        pmk_ctx.h0 ^= temp_ctx.h0; pmk_ctx.h1 ^= temp_ctx.h1;
        pmk_ctx.h2 ^= temp_ctx.h2; pmk_ctx.h3 ^= temp_ctx.h3;
        pmk_ctx.h4 ^= temp_ctx.h4;
    }

    CPY_DEVCTX(pmk_ctx, outbuffer[idx].pmk2);
}


/* Custom version of pbkdf2: 
 * - Works on a list of passwords 
 * - Outputs a list of 32byte derived keys
 * - num_pwds must be multiple of 64
 */
int cuda_pbkdf2_hmac_sha1_32(unsigned char **pwdlst, int num_pwds, unsigned char *salt, 
						  size_t saltlen, uint32_t iterations, uint8_t **key){


	unsigned char pad[64], temp[32], *passwd;
	int i=0,j=0,passwdlen;
	SHA_CTX ctx_pad;
    gpu_inbuffer *h_inbuffer, *d_inbuffer;
    gpu_outbuffer *h_outbuffer, *d_outbuffer;
	hipError_t cudaReturnValue;
	
	/* cuda allocation */
	h_inbuffer = (gpu_inbuffer *)calloc(num_pwds, sizeof(gpu_inbuffer));
	if(h_inbuffer == NULL){
		errprint("Malloc error\n");
		return 0;
	}

	h_outbuffer = (gpu_outbuffer *)calloc(num_pwds, sizeof(gpu_outbuffer));
	if(h_outbuffer == NULL){
		errprint("Malloc error\n");
		return 0;
	}

	cudaReturnValue = hipMalloc((void **) &d_inbuffer, (num_pwds) * sizeof(gpu_inbuffer));
	if(cudaReturnValue != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
		return 0;
	}

	cudaReturnValue = hipMalloc((void **) &d_outbuffer, (num_pwds) * sizeof(gpu_outbuffer));
	if(cudaReturnValue != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
		return 0;
	}

	for(i = 0; i < num_pwds; i++){
		passwd = pwdlst[i];
		passwdlen = strlen((const char *)passwd);

		memcpy(pad, passwd, passwdlen);
        memset(pad + passwdlen, 0, sizeof(pad) - passwdlen);

        for (j = 0; j < 16; j++)
            ((unsigned int*)pad)[j] ^= 0x36363636;

        SHA1_Init(&ctx_pad);
        SHA1_Update(&ctx_pad, pad, sizeof(pad));
        CPY_DEVCTX(ctx_pad, h_inbuffer[i].ctx_ipad);

        for (j = 0; j < 16; j++)
            ((unsigned int*)pad)[j] ^= 0x6a6a6a6a;

        SHA1_Init(&ctx_pad);
        SHA1_Update(&ctx_pad, pad, sizeof(pad));
        CPY_DEVCTX(ctx_pad, h_inbuffer[i].ctx_opad);

        salt[saltlen + 4 - 1] = '\1';
        HMAC(EVP_sha1(), passwd, passwdlen, salt, saltlen + 4, temp, NULL);
        GET_BE(h_inbuffer[i].e1.h0, temp, 0);
        GET_BE(h_inbuffer[i].e1.h1, temp, 4);
        GET_BE(h_inbuffer[i].e1.h2, temp, 8);
        GET_BE(h_inbuffer[i].e1.h3, temp, 12);
        GET_BE(h_inbuffer[i].e1.h4, temp, 16);

        salt[saltlen + 4 - 1] = '\2';
        HMAC(EVP_sha1(), passwd, passwdlen, salt, saltlen + 4, temp, NULL);
        GET_BE(h_inbuffer[i].e2.h0, temp, 0);
        GET_BE(h_inbuffer[i].e2.h1, temp, 4);
        GET_BE(h_inbuffer[i].e2.h2, temp, 8);
        GET_BE(h_inbuffer[i].e2.h3, temp, 12);
        GET_BE(h_inbuffer[i].e2.h4, temp, 16);
	}

	cudaReturnValue = hipMemcpy(d_inbuffer, h_inbuffer, num_pwds * sizeof(gpu_inbuffer), hipMemcpyHostToDevice);
	if(cudaReturnValue != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
		return 0;
	}


	/* call the cuda kernel */
	kernel_pbkdf2_sha1_32<<<num_pwds/64, 64>>>(d_inbuffer, d_outbuffer, iterations);
	hipDeviceSynchronize();

	if((cudaReturnValue = hipGetLastError()) != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
        return 0;
    }

	cudaReturnValue = hipMemcpy(h_outbuffer, d_outbuffer, num_pwds * sizeof(gpu_outbuffer), hipMemcpyDeviceToHost);
	if(cudaReturnValue != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
		return 0;
	}
	
	for(i=0;i<num_pwds;i++){
	    PUT_BE(h_outbuffer[0].pmk1.h0, temp,  0); PUT_BE(h_outbuffer[0].pmk1.h1, temp,  4);
	    PUT_BE(h_outbuffer[0].pmk1.h2, temp,  8); PUT_BE(h_outbuffer[0].pmk1.h3, temp, 12);
	    PUT_BE(h_outbuffer[0].pmk1.h4, temp, 16); PUT_BE(h_outbuffer[0].pmk2.h0, temp, 20);
	    PUT_BE(h_outbuffer[0].pmk2.h1, temp, 24); PUT_BE(h_outbuffer[0].pmk2.h2, temp, 28);
		memcpy(key[i], temp, 32);
	}

	cudaReturnValue = hipFree(d_inbuffer);
	if(cudaReturnValue != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
		return 0;
	}

	cudaReturnValue = hipFree(d_outbuffer);
	if(cudaReturnValue != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
		return 0;
	}

	free(h_outbuffer);
	free(h_inbuffer);

	return 1;

}
