#include "hip/hip_runtime.h"
extern "C"{
#include "../utils.h"
#include "cuda_pbkdf2.h"
}
#include <openssl/hmac.h>
#include <openssl/sha.h>
#include <string.h>
#include <hip/hip_runtime.h>

__device__
void sha256_process( const SHA256_DEV_CTX *ctx, SHA256_DEV_CTX *data){

	uint32_t temp1, temp2, W[64], A, B, C, D, E, F, G, H, i;

	W[ 0] = data->h0;
	W[ 1] = data->h1;
	W[ 2] = data->h2;
	W[ 3] = data->h3;
	W[ 4] = data->h4;
	W[ 5] = data->h5;
	W[ 6] = data->h6;
	W[ 7] = data->h7;
	W[ 8] = 0x80000000;
	W[ 9] = 0;
	W[10] = 0;
	W[11] = 0;
	W[12] = 0;
	W[13] = 0;
	W[14] = 0;
	W[15] = (64+32)*8;

	A = ctx->h0;
	B = ctx->h1;
	C = ctx->h2;
	D = ctx->h3;
	E = ctx->h4;
	F = ctx->h5;
	G = ctx->h6;
	H = ctx->h7;

#undef RS
#define RS(x,n) (x >> n)

#undef RR
#define RR(x,n) ((x >> n) | (x << (32 - n)))

#undef R
#define R(t)																	\
(																				\
	temp1 = RR( W[(t - 15)],  7) ^ RR( W[(t - 15)], 18) ^			\
			RS( W[(t - 15)],  3),										\
	temp2 = RR( W[(t -  2)], 17) ^ RR( W[(t -  2)], 19) ^			\
			RS( W[(t -  2)], 10),										\
	( W[t] = W[(t - 16)] + temp1 + W[(t -  7)] + temp2 )	\
)

#undef S1
#define S1(x) (RR(x,6) ^ RR(x,11) ^ RR(x,25))

#undef S0
#define S0(x) (RR(x,2) ^ RR(x,13) ^ RR(x,22))

#undef maj
#define maj(x,y,z) ((x & y) ^ (x & z) ^ (y & z))

#undef ch
#define ch(x,y,z) (((x) & (y)) ^ ((~(x)) & (z)))

#undef P
#define P(a,b,c,d,e,f,g,h,x,K)					\
{												\
	temp1 = h + S1(e) + ch(e,f,g) + K + x;		\
	temp2 = S0(a) + maj(a,b,c);					\
	d += temp1;									\
	h = temp1 + temp2;							\
}

	for(i=16;i<64;i++){
		R(i);
	}

	P( A, B, C, D, E, F, G, H, W[ 0], 0x428a2f98 );
	P( H, A, B, C, D, E, F, G, W[ 1], 0x71374491 );
	P( G, H, A, B, C, D, E, F, W[ 2], 0xb5c0fbcf );
	P( F, G, H, A, B, C, D, E, W[ 3], 0xe9b5dba5 );
	P( E, F, G, H, A, B, C, D, W[ 4], 0x3956c25b );
	P( D, E, F, G, H, A, B, C, W[ 5], 0x59f111f1 );
	P( C, D, E, F, G, H, A, B, W[ 6], 0x923f82a4 );
	P( B, C, D, E, F, G, H, A, W[ 7], 0xab1c5ed5 );
	P( A, B, C, D, E, F, G, H, W[ 8], 0xd807aa98 );
	P( H, A, B, C, D, E, F, G, W[ 9], 0x12835b01 );
	P( G, H, A, B, C, D, E, F, W[10], 0x243185be );
	P( F, G, H, A, B, C, D, E, W[11], 0x550c7dc3 );
	P( E, F, G, H, A, B, C, D, W[12], 0x72be5d74 );
	P( D, E, F, G, H, A, B, C, W[13], 0x80deb1fe );
	P( C, D, E, F, G, H, A, B, W[14], 0x9bdc06a7 );
	P( B, C, D, E, F, G, H, A, W[15], 0xc19bf174 );
	P( A, B, C, D, E, F, G, H, W[16], 0xe49b69c1 );
	P( H, A, B, C, D, E, F, G, W[17], 0xefbe4786 );
	P( G, H, A, B, C, D, E, F, W[18], 0x0fc19dc6 );
	P( F, G, H, A, B, C, D, E, W[19], 0x240ca1cc );
	P( E, F, G, H, A, B, C, D, W[20], 0x2de92c6f );
	P( D, E, F, G, H, A, B, C, W[21], 0x4a7484aa );
	P( C, D, E, F, G, H, A, B, W[22], 0x5cb0a9dc );
	P( B, C, D, E, F, G, H, A, W[23], 0x76f988da );
	P( A, B, C, D, E, F, G, H, W[24], 0x983e5152 );
	P( H, A, B, C, D, E, F, G, W[25], 0xa831c66d );
	P( G, H, A, B, C, D, E, F, W[26], 0xb00327c8 );
	P( F, G, H, A, B, C, D, E, W[27], 0xbf597fc7 );
	P( E, F, G, H, A, B, C, D, W[28], 0xc6e00bf3 );
	P( D, E, F, G, H, A, B, C, W[29], 0xd5a79147 );
	P( C, D, E, F, G, H, A, B, W[30], 0x06ca6351 );
	P( B, C, D, E, F, G, H, A, W[31], 0x14292967 );
	P( A, B, C, D, E, F, G, H, W[32], 0x27b70a85 );
	P( H, A, B, C, D, E, F, G, W[33], 0x2e1b2138 );
	P( G, H, A, B, C, D, E, F, W[34], 0x4d2c6dfc );
	P( F, G, H, A, B, C, D, E, W[35], 0x53380d13 );
	P( E, F, G, H, A, B, C, D, W[36], 0x650a7354 );
	P( D, E, F, G, H, A, B, C, W[37], 0x766a0abb );
	P( C, D, E, F, G, H, A, B, W[38], 0x81c2c92e );
	P( B, C, D, E, F, G, H, A, W[39], 0x92722c85 );
	P( A, B, C, D, E, F, G, H, W[40], 0xa2bfe8a1 );
	P( H, A, B, C, D, E, F, G, W[41], 0xa81a664b );
	P( G, H, A, B, C, D, E, F, W[42], 0xc24b8b70 );
	P( F, G, H, A, B, C, D, E, W[43], 0xc76c51a3 );
	P( E, F, G, H, A, B, C, D, W[44], 0xd192e819 );
	P( D, E, F, G, H, A, B, C, W[45], 0xd6990624 );
	P( C, D, E, F, G, H, A, B, W[46], 0xf40e3585 );
	P( B, C, D, E, F, G, H, A, W[47], 0x106aa070 );
	P( A, B, C, D, E, F, G, H, W[48], 0x19a4c116 );
	P( H, A, B, C, D, E, F, G, W[49], 0x1e376c08 );
	P( G, H, A, B, C, D, E, F, W[50], 0x2748774c );
	P( F, G, H, A, B, C, D, E, W[51], 0x34b0bcb5 );
	P( E, F, G, H, A, B, C, D, W[52], 0x391c0cb3 );
	P( D, E, F, G, H, A, B, C, W[53], 0x4ed8aa4a );
	P( C, D, E, F, G, H, A, B, W[54], 0x5b9cca4f );
	P( B, C, D, E, F, G, H, A, W[55], 0x682e6ff3 );
	P( A, B, C, D, E, F, G, H, W[56], 0x748f82ee );
	P( H, A, B, C, D, E, F, G, W[57], 0x78a5636f );
	P( G, H, A, B, C, D, E, F, W[58], 0x84c87814 );
	P( F, G, H, A, B, C, D, E, W[59], 0x8cc70208 );
	P( E, F, G, H, A, B, C, D, W[60], 0x90befffa );
	P( D, E, F, G, H, A, B, C, W[61], 0xa4506ceb );
	P( C, D, E, F, G, H, A, B, W[62], 0xbef9a3f7 );
	P( B, C, D, E, F, G, H, A, W[63], 0xc67178f2 );

	data->h0 = ctx->h0 + A;
	data->h1 = ctx->h1 + B;
	data->h2 = ctx->h2 + C;
	data->h3 = ctx->h3 + D;
	data->h4 = ctx->h4 + E;
	data->h5 = ctx->h5 + F;
	data->h6 = ctx->h6 + G;
	data->h7 = ctx->h7 + H;
}

__global__ void kernel_pbkdf2_sha256_32( gpu_inbuffer256 *inbuffer, 
				gpu_outbuffer256 *outbuffer, int *iterations, int num_pwds) {

    int i;
	SHA256_DEV_CTX temp_ctx, pmk_ctx;

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx<num_pwds){
		    CPY_DEVCTX2(inbuffer[idx].e1, temp_ctx);
		    CPY_DEVCTX2(temp_ctx, pmk_ctx);

		    for( i = 0; i < iterations[idx]-1; i++ ){
		    	sha256_process( &inbuffer[idx].ctx_ipad, &temp_ctx);
		    	sha256_process( &inbuffer[idx].ctx_opad, &temp_ctx);
		        pmk_ctx.h0 ^= temp_ctx.h0; pmk_ctx.h1 ^= temp_ctx.h1;
		        pmk_ctx.h2 ^= temp_ctx.h2; pmk_ctx.h3 ^= temp_ctx.h3;
		        pmk_ctx.h4 ^= temp_ctx.h4; pmk_ctx.h5 ^= temp_ctx.h5;
				pmk_ctx.h6 ^= temp_ctx.h6; pmk_ctx.h7 ^= temp_ctx.h7;
		    }
		    CPY_DEVCTX2(pmk_ctx, outbuffer[idx].pmk);
	}
}


extern "C"{
int cuda_pbkdf2_hmac_sha256_32(unsigned char **pwdlst, size_t num_pwds, 
				unsigned char *salt, size_t saltlen, uint32_t iterations, 
				uint8_t **key){


	unsigned char pad[64], temp[32], *passwd;
	int i=0, j=0, passwdlen,r=1, *d_iter, blks;
	SHA256_CTX ctx_pad;
    gpu_inbuffer256 *h_inbuffer, *d_inbuffer;
    gpu_outbuffer256 *h_outbuffer, *d_outbuffer;
	hipError_t cudaReturnValue;

	h_inbuffer = (gpu_inbuffer256 *)calloc(num_pwds, sizeof(gpu_inbuffer256));
	if(h_inbuffer == NULL){
		errprint("Malloc error\n");
		return 0;
	}

	h_outbuffer = (gpu_outbuffer256 *)calloc(num_pwds, sizeof(gpu_outbuffer256));
	if(h_outbuffer == NULL){
		errprint("Malloc error\n");
		return 0;
	}

	cudaReturnValue = hipMalloc((void **) &d_iter, (num_pwds) * sizeof(int));
	if(cudaReturnValue != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
		return 0;
	}

	cudaReturnValue = hipMalloc((void **) &d_inbuffer, (num_pwds) * sizeof(gpu_inbuffer256));
	if(cudaReturnValue != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
		return 0;
	}

	cudaReturnValue = hipMalloc((void **) &d_outbuffer, (num_pwds) * sizeof(gpu_outbuffer256));
	if(cudaReturnValue != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
		return 0;
	}

	for(i = 0; i < num_pwds; i++){

		cudaReturnValue = hipMemcpy(&d_iter[i], &iterations, sizeof(int), hipMemcpyHostToDevice);
		if(cudaReturnValue != hipSuccess){
			errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));

			r=0;
			goto end;
		}

		passwd = pwdlst[i];
		passwdlen = strlen((const char *)passwd);

		memcpy(pad, passwd, passwdlen);
        memset(pad + passwdlen, 0, sizeof(pad) - passwdlen);

        for (j = 0; j < 16; j++)
            ((unsigned int*)pad)[j] ^= 0x36363636;

        SHA256_Init(&ctx_pad);
        SHA256_Update(&ctx_pad, pad, sizeof(pad));
        CPY_DEVCTX2_openSSL(ctx_pad, h_inbuffer[i].ctx_ipad);

        for (j = 0; j < 16; j++)
            ((unsigned int*)pad)[j] ^= 0x6a6a6a6a;

        SHA256_Init(&ctx_pad);
        SHA256_Update(&ctx_pad, pad, sizeof(pad));
        CPY_DEVCTX2_openSSL(ctx_pad, h_inbuffer[i].ctx_opad);

        salt[saltlen + 4 - 1] = '\1';
        HMAC(EVP_sha256(), passwd, passwdlen, salt, saltlen + 4, temp, NULL);
        GET_BE(h_inbuffer[i].e1.h0, temp, 0);
        GET_BE(h_inbuffer[i].e1.h1, temp, 4);
        GET_BE(h_inbuffer[i].e1.h2, temp, 8);
        GET_BE(h_inbuffer[i].e1.h3, temp, 12);
        GET_BE(h_inbuffer[i].e1.h4, temp, 16);
        GET_BE(h_inbuffer[i].e1.h5, temp, 20);
        GET_BE(h_inbuffer[i].e1.h6, temp, 24);
        GET_BE(h_inbuffer[i].e1.h7, temp, 28);

	}

	cudaReturnValue = hipMemcpy(d_inbuffer, h_inbuffer, num_pwds * sizeof(gpu_inbuffer256), hipMemcpyHostToDevice);
	if(cudaReturnValue != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
		r=0;
		goto end;
	}

	blks = ceil((num_pwds/64));

	kernel_pbkdf2_sha256_32<<<blks, 64>>>(d_inbuffer, d_outbuffer, d_iter, num_pwds);
	hipDeviceSynchronize();

	if((cudaReturnValue = hipGetLastError()) != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
		r=0;
		goto end;
    }

	cudaReturnValue = hipMemcpy(h_outbuffer, d_outbuffer, num_pwds * sizeof(gpu_outbuffer256), hipMemcpyDeviceToHost);
	if(cudaReturnValue != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
		r=0;
		goto end;
	}
	
	for(i=0;i<num_pwds;i++){
	    PUT_BE(h_outbuffer[i].pmk.h0, temp,  0); 
		PUT_BE(h_outbuffer[i].pmk.h1, temp,  4);
		PUT_BE(h_outbuffer[i].pmk.h2, temp,  8); 
		PUT_BE(h_outbuffer[i].pmk.h3, temp, 12);
		PUT_BE(h_outbuffer[i].pmk.h4, temp, 16); 
		PUT_BE(h_outbuffer[i].pmk.h5, temp, 20);
	    PUT_BE(h_outbuffer[i].pmk.h6, temp, 24); 
		PUT_BE(h_outbuffer[i].pmk.h7, temp, 28);
		memcpy(key[i], temp, 32);
	}

end:

	cudaReturnValue = hipFree(d_inbuffer);
	if(cudaReturnValue != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
		return 0;
	}

	cudaReturnValue = hipFree(d_outbuffer);
	if(cudaReturnValue != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
		r=0;
		return 0;
	}

	free(h_outbuffer);
	free(h_inbuffer);

	return r;

}
}/* end of extern "C"{ */

