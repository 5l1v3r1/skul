#include "hip/hip_runtime.h"
extern "C"{
#include "../utils.h"
#include "cuda_pbkdf2.h"
}
#include <openssl/hmac.h>
#include <openssl/sha.h>
#include <string.h>
#include <hip/hip_runtime.h>

__device__
void sha512_process( const SHA512_DEV_CTX *ctx, SHA512_DEV_CTX *data){

	uint64_t temp1, temp2, W[16], A, B, C, D, E, F, G, H;

	W[ 0] = data->h0;
	W[ 1] = data->h1;
	W[ 2] = data->h2;
	W[ 3] = data->h3;
	W[ 4] = data->h4;
	W[ 5] = data->h5;
	W[ 6] = data->h6;
	W[ 7] = data->h7;
	W[ 8] = 0x8000000000000000;
	W[ 9] = 0;
	W[10] = 0;
	W[11] = 0;
	W[12] = 0;
	W[13] = 0;
	W[14] = 0;
	W[15] = (128+64)*8;

	A = ctx->h0;
	B = ctx->h1;
	C = ctx->h2;
	D = ctx->h3;
	E = ctx->h4;
	F = ctx->h5;
	G = ctx->h6;
	H = ctx->h7;


#undef RS
#define RS(x,n) (x >> n)

#undef RR
#define RR(x,n) ((x >> n) | (x << (64 - n)))

#undef R
#define R(t)																	\
(																				\
	temp1 = RR( W[(t - 15) & 0x0F],  1) ^ RR( W[(t - 15) & 0x0F], 8) ^			\
			RS( W[(t - 15) & 0x0F],  7),										\
	temp2 = RR( W[(t -  2) & 0x0F], 19) ^ RR( W[(t -  2) & 0x0F], 61) ^			\
			RS( W[(t -  2) & 0x0F], 6),											\
	( W[t & 0x0F] = W[(t - 16) & 0x0F] + temp1 + W[(t -  7) & 0x0F] + temp2 )	\
)

#undef S1
#define S1(x) (RR(x,14) ^ RR(x,18) ^ RR(x,41))

#undef S0
#define S0(x) (RR(x,28) ^ RR(x,34) ^ RR(x,39))

#undef maj
#define maj(x,y,z) ((x & y) ^ (x & z) ^ (y & z))

#undef ch
#define ch(x,y,z) (((x) & (y)) ^ ((~(x)) & (z)))

#undef P
#define P(a,b,c,d,e,f,g,h,x,K)					\
{												\
	temp1 = h + S1(e) + ch(e,f,g) + K + x;		\
	temp2 = S0(a) + maj(a,b,c);					\
	d += temp1;									\
	h = temp1 + temp2;							\
}


	P( A, B, C, D, E, F, G, H, W[0] , 0x428a2f98d728ae22 );
	P( H, A, B, C, D, E, F, G, W[1] , 0x7137449123ef65cd );
	P( G, H, A, B, C, D, E, F, W[2] , 0xb5c0fbcfec4d3b2f );
	P( F, G, H, A, B, C, D, E, W[3] , 0xe9b5dba58189dbbc );
	P( E, F, G, H, A, B, C, D, W[4] , 0x3956c25bf348b538 );
	P( D, E, F, G, H, A, B, C, W[5] , 0x59f111f1b605d019 );
	P( C, D, E, F, G, H, A, B, W[6] , 0x923f82a4af194f9b );
	P( B, C, D, E, F, G, H, A, W[7] , 0xab1c5ed5da6d8118 );
	P( A, B, C, D, E, F, G, H, W[8] , 0xd807aa98a3030242 );
	P( H, A, B, C, D, E, F, G, W[9] , 0x12835b0145706fbe );
	P( G, H, A, B, C, D, E, F, W[10], 0x243185be4ee4b28c );
	P( F, G, H, A, B, C, D, E, W[11], 0x550c7dc3d5ffb4e2 );
	P( E, F, G, H, A, B, C, D, W[12], 0x72be5d74f27b896f );
	P( D, E, F, G, H, A, B, C, W[13], 0x80deb1fe3b1696b1 );
	P( C, D, E, F, G, H, A, B, W[14], 0x9bdc06a725c71235 );
	P( B, C, D, E, F, G, H, A, W[15], 0xc19bf174cf692694 );
	P( A, B, C, D, E, F, G, H, R(16), 0xe49b69c19ef14ad2 );
	P( H, A, B, C, D, E, F, G, R(17), 0xefbe4786384f25e3 );
	P( G, H, A, B, C, D, E, F, R(18), 0x0fc19dc68b8cd5b5 );
	P( F, G, H, A, B, C, D, E, R(19), 0x240ca1cc77ac9c65 );
	P( E, F, G, H, A, B, C, D, R(20), 0x2de92c6f592b0275 );
	P( D, E, F, G, H, A, B, C, R(21), 0x4a7484aa6ea6e483 );
	P( C, D, E, F, G, H, A, B, R(22), 0x5cb0a9dcbd41fbd4 );
	P( B, C, D, E, F, G, H, A, R(23), 0x76f988da831153b5 );
	P( A, B, C, D, E, F, G, H, R(24), 0x983e5152ee66dfab );
	P( H, A, B, C, D, E, F, G, R(25), 0xa831c66d2db43210 );
	P( G, H, A, B, C, D, E, F, R(26), 0xb00327c898fb213f );
	P( F, G, H, A, B, C, D, E, R(27), 0xbf597fc7beef0ee4 );
	P( E, F, G, H, A, B, C, D, R(28), 0xc6e00bf33da88fc2 );
	P( D, E, F, G, H, A, B, C, R(29), 0xd5a79147930aa725 );
	P( C, D, E, F, G, H, A, B, R(30), 0x06ca6351e003826f );
	P( B, C, D, E, F, G, H, A, R(31), 0x142929670a0e6e70 );
	P( A, B, C, D, E, F, G, H, R(32), 0x27b70a8546d22ffc );
	P( H, A, B, C, D, E, F, G, R(33), 0x2e1b21385c26c926 );
	P( G, H, A, B, C, D, E, F, R(34), 0x4d2c6dfc5ac42aed );
	P( F, G, H, A, B, C, D, E, R(35), 0x53380d139d95b3df );
	P( E, F, G, H, A, B, C, D, R(36), 0x650a73548baf63de );
	P( D, E, F, G, H, A, B, C, R(37), 0x766a0abb3c77b2a8 );
	P( C, D, E, F, G, H, A, B, R(38), 0x81c2c92e47edaee6 );
	P( B, C, D, E, F, G, H, A, R(39), 0x92722c851482353b );
	P( A, B, C, D, E, F, G, H, R(40), 0xa2bfe8a14cf10364 );
	P( H, A, B, C, D, E, F, G, R(41), 0xa81a664bbc423001 );
	P( G, H, A, B, C, D, E, F, R(42), 0xc24b8b70d0f89791 );
	P( F, G, H, A, B, C, D, E, R(43), 0xc76c51a30654be30 );
	P( E, F, G, H, A, B, C, D, R(44), 0xd192e819d6ef5218 );
	P( D, E, F, G, H, A, B, C, R(45), 0xd69906245565a910 );
	P( C, D, E, F, G, H, A, B, R(46), 0xf40e35855771202a );
	P( B, C, D, E, F, G, H, A, R(47), 0x106aa07032bbd1b8 );
	P( A, B, C, D, E, F, G, H, R(48), 0x19a4c116b8d2d0c8 );
	P( H, A, B, C, D, E, F, G, R(49), 0x1e376c085141ab53 );
	P( G, H, A, B, C, D, E, F, R(50), 0x2748774cdf8eeb99 );
	P( F, G, H, A, B, C, D, E, R(51), 0x34b0bcb5e19b48a8 );
	P( E, F, G, H, A, B, C, D, R(52), 0x391c0cb3c5c95a63 );
	P( D, E, F, G, H, A, B, C, R(53), 0x4ed8aa4ae3418acb );
	P( C, D, E, F, G, H, A, B, R(54), 0x5b9cca4f7763e373 );
	P( B, C, D, E, F, G, H, A, R(55), 0x682e6ff3d6b2b8a3 );
	P( A, B, C, D, E, F, G, H, R(56), 0x748f82ee5defb2fc );
	P( H, A, B, C, D, E, F, G, R(57), 0x78a5636f43172f60 );
	P( G, H, A, B, C, D, E, F, R(58), 0x84c87814a1f0ab72 );
	P( F, G, H, A, B, C, D, E, R(59), 0x8cc702081a6439ec );
	P( E, F, G, H, A, B, C, D, R(60), 0x90befffa23631e28 );
	P( D, E, F, G, H, A, B, C, R(61), 0xa4506cebde82bde9 );
	P( C, D, E, F, G, H, A, B, R(62), 0xbef9a3f7b2c67915 );
	P( B, C, D, E, F, G, H, A, R(63), 0xc67178f2e372532b );
    P( A, B, C, D, E, F, G, H, R(64), 0xca273eceea26619c );
    P( H, A, B, C, D, E, F, G, R(65), 0xd186b8c721c0c207 ); 
    P( G, H, A, B, C, D, E, F, R(66), 0xeada7dd6cde0eb1e );
    P( F, G, H, A, B, C, D, E, R(67), 0xf57d4f7fee6ed178 );
    P( E, F, G, H, A, B, C, D, R(68), 0x06f067aa72176fba );
    P( D, E, F, G, H, A, B, C, R(69), 0x0a637dc5a2c898a6 );
    P( C, D, E, F, G, H, A, B, R(70), 0x113f9804bef90dae );
    P( B, C, D, E, F, G, H, A, R(71), 0x1b710b35131c471b );
	P( A, B, C, D, E, F, G, H, R(72), 0x28db77f523047d84 );
    P( H, A, B, C, D, E, F, G, R(73), 0x32caab7b40c72493 );
    P( G, H, A, B, C, D, E, F, R(74), 0x3c9ebe0a15c9bebc );
    P( F, G, H, A, B, C, D, E, R(75), 0x431d67c49c100d4c );
    P( E, F, G, H, A, B, C, D, R(76), 0x4cc5d4becb3e42b6 );
    P( D, E, F, G, H, A, B, C, R(77), 0x597f299cfc657e2a );
    P( C, D, E, F, G, H, A, B, R(78), 0x5fcb6fab3ad6faec );
    P( B, C, D, E, F, G, H, A, R(79), 0x6c44198c4a475817 );
                                    
                                    
	data->h0 = ctx->h0 + A;
	data->h1 = ctx->h1 + B;
	data->h2 = ctx->h2 + C;
	data->h3 = ctx->h3 + D;
	data->h4 = ctx->h4 + E;
	data->h5 = ctx->h5 + F;
	data->h6 = ctx->h6 + G;
	data->h7 = ctx->h7 + H;

}


__global__ void kernel_pbkdf2_sha512_32( gpu_inbuffer512 *inbuffer, 
									gpu_outbuffer512 *outbuffer, int *iterations, int num_pwds) {
	
	int i;
	SHA512_DEV_CTX temp_ctx, pmk_ctx; 
    
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;  
	if(idx<num_pwds){

	    CPY_DEVCTX2(inbuffer[idx].e1, temp_ctx);
	    CPY_DEVCTX2(temp_ctx, pmk_ctx);
	
	    for( i = 0; i < iterations[idx]-1; i++ ){
	        sha512_process( &inbuffer[idx].ctx_ipad, &temp_ctx);
	        sha512_process( &inbuffer[idx].ctx_opad, &temp_ctx);
	        pmk_ctx.h0 ^= temp_ctx.h0; pmk_ctx.h1 ^= temp_ctx.h1;
	        pmk_ctx.h2 ^= temp_ctx.h2; pmk_ctx.h3 ^= temp_ctx.h3;
	        pmk_ctx.h4 ^= temp_ctx.h4; pmk_ctx.h5 ^= temp_ctx.h5;
			pmk_ctx.h6 ^= temp_ctx.h6; pmk_ctx.h7 ^= temp_ctx.h7;
	    }
	    CPY_DEVCTX2(pmk_ctx, outbuffer[idx].pmk);
	}
}


/* Custom version of pbkdf2_hmac_sha512: 
 * - Works on a list of passwords 
 * - Outputs a list of 32byte derived keys
 */
extern "C"{
int cuda_pbkdf2_hmac_sha512_32(unsigned char **pwdlst, size_t num_pwds, unsigned char *salt, 
						  size_t saltlen, uint32_t iterations, uint8_t **key){


	unsigned char pad[128], temp[64], *passwd;
	size_t i=0;
	int j=0, passwdlen,r=1, *d_iter, blks;
	SHA512_CTX ctx_pad;
    gpu_inbuffer512 *h_inbuffer, *d_inbuffer;
    gpu_outbuffer512 *h_outbuffer, *d_outbuffer;
	hipError_t cudaReturnValue;

	/* cuda allocation */
	h_inbuffer = (gpu_inbuffer512 *)calloc(num_pwds, sizeof(gpu_inbuffer512));
	if(h_inbuffer == NULL){
		errprint("Malloc error\n");
		return 0;
	}

	h_outbuffer = (gpu_outbuffer512 *)calloc(num_pwds, sizeof(gpu_outbuffer512));
	if(h_outbuffer == NULL){
		errprint("Malloc error\n");
		return 0;
	}

	cudaReturnValue = hipMalloc((void **) &d_iter, (num_pwds) * sizeof(int));
	if(cudaReturnValue != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
		return 0;
	}

	cudaReturnValue = hipMalloc((void **) &d_inbuffer, (num_pwds) * sizeof(gpu_inbuffer512));
	if(cudaReturnValue != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
		return 0;
	}

	cudaReturnValue = hipMalloc((void **) &d_outbuffer, (num_pwds) * sizeof(gpu_outbuffer512));
	if(cudaReturnValue != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
		return 0;
	}

	for(i = 0; i < num_pwds; i++){

		cudaReturnValue = hipMemcpy(&d_iter[i], &iterations, sizeof(int), hipMemcpyHostToDevice);
		if(cudaReturnValue != hipSuccess){
			errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));

			r=0;
			goto end;
		}

		passwd = pwdlst[i];
		passwdlen = strlen((const char *)passwd);

		memcpy(pad, passwd, passwdlen);
        memset(pad + passwdlen, 0, sizeof(pad) - passwdlen);

        for (j = 0; j < 32; j++)
            ((unsigned int*)pad)[j] ^= 0x36363636;
        SHA512_Init(&ctx_pad);
        SHA512_Update(&ctx_pad, pad, sizeof(pad));
        CPY_DEVCTX2_openSSL(ctx_pad, h_inbuffer[i].ctx_ipad);

        for (j = 0; j < 32; j++)
            ((unsigned int*)pad)[j] ^= 0x6a6a6a6a;
        SHA512_Init(&ctx_pad);
        SHA512_Update(&ctx_pad, pad, sizeof(pad));
        CPY_DEVCTX2_openSSL(ctx_pad, h_inbuffer[i].ctx_opad);

        salt[saltlen + 4 - 1] = '\1';
        HMAC(EVP_sha512(), passwd, passwdlen, salt, saltlen + 4, temp, NULL);
        GET_BE64(h_inbuffer[i].e1.h0, temp, 0);
        GET_BE64(h_inbuffer[i].e1.h1, temp, 8);
        GET_BE64(h_inbuffer[i].e1.h2, temp, 16);
        GET_BE64(h_inbuffer[i].e1.h3, temp, 24);
        GET_BE64(h_inbuffer[i].e1.h4, temp, 32);
        GET_BE64(h_inbuffer[i].e1.h5, temp, 40);
        GET_BE64(h_inbuffer[i].e1.h6, temp, 48);
        GET_BE64(h_inbuffer[i].e1.h7, temp, 56);
	}

	cudaReturnValue = hipMemcpy(d_inbuffer, h_inbuffer, num_pwds * sizeof(gpu_inbuffer512), hipMemcpyHostToDevice);
	if(cudaReturnValue != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
		r=0;
		goto end;
	}

	blks = ceil((num_pwds/64));

	kernel_pbkdf2_sha512_32<<<blks, 64>>>(d_inbuffer, d_outbuffer, d_iter, num_pwds);
	hipDeviceSynchronize();

	if((cudaReturnValue = hipGetLastError()) != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
		r=0;
		goto end;
    }

	cudaReturnValue = hipMemcpy(h_outbuffer, d_outbuffer, num_pwds * sizeof(gpu_outbuffer512), hipMemcpyDeviceToHost);
	if(cudaReturnValue != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
		r=0;
		goto end;
	}
	
	for(i=0;i<num_pwds;i++){
	    PUT_BE64(h_outbuffer[i].pmk.h0, temp,  0);  PUT_BE64(h_outbuffer[i].pmk.h1, temp,  8);
	    PUT_BE64(h_outbuffer[i].pmk.h2, temp,  16); PUT_BE64(h_outbuffer[i].pmk.h3, temp, 24);
		memcpy(key[i], temp, 32);
	}

end:

	cudaReturnValue = hipFree(d_inbuffer);
	if(cudaReturnValue != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
		return 0;
	}

	cudaReturnValue = hipFree(d_outbuffer);
	if(cudaReturnValue != hipSuccess){
		errprint("Cuda error: %d - %s\n",cudaReturnValue, hipGetErrorString(cudaReturnValue));
		r=0;
		return 0;
	}

	free(h_outbuffer);
	free(h_inbuffer);

	return r;

}
}/* end of extern "C"{ */
